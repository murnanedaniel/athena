#include "hip/hip_runtime.h"
#include <iostream>
#include "ISF_FastCaloGpu/GeoLoadGpu.h"

__global__ void testHello() { printf( "Hello, I am from GPU thread %d\n", threadIdx.x ); }

/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

__global__ void testCell( CaloDetDescrElement_Gpu* cells, unsigned long index ) {
  CaloDetDescrElement_Gpu* cell   = &cells[index];
  int                  sample = cell->getSampling();
  float                eta    = cell->eta();
  float                phi    = cell->phi();

  long long hashid = cell->calo_hash();

  printf( " From GPU cell index %ld , hashid=%ld, eta=%f, phi=%f, sample=%d \n", index, hashid, eta, phi, sample );
}

/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

__global__ void testGeo( CaloDetDescrElement_Gpu* cells, GeoRegion* regions, unsigned int nregions, unsigned long ncells,
                         int r, int ir, int ip ) {

  int                neta  = regions[r].cell_grid_eta();
  int                nphi  = regions[r].cell_grid_phi();
  unsigned long long index = regions[r].cell_grid_g()[ir * nphi + ip];
  printf( " From GPU.., region %d, cell_grid[%d][%d]: [%d][%d] index=%lu \n", r, ir, ip, neta, nphi, index );

  CaloDetDescrElement_Gpu* c = &cells[index];

  long long hashid = c->calo_hash();
  long long id     = c->identify();
  int       sample = c->getSampling();
  float     eta    = c->eta();
  float     phi    = c->phi();

  printf( " From GPU.., region %d, cell_grid[%d][%d]: index %lu index, hashid=%ld,eta=%f, phi=%f, sample=%d , ID=%ld "
          "cell_ptr=%#015lx \n",
          r, ir, ip, index, hashid, eta, phi, sample, id, regions[r].all_cells() );

  CaloDetDescrElement_Gpu cc = ( regions[r].all_cells() )[index];
  printf( " GPU test region have cells: cell index %lu, eta=%f phi=%f size of cell*GPU=%lu\n", index, cc.eta(),
          cc.phi(), sizeof( CaloDetDescrElement_Gpu* ) );
}

/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

__global__ void testGeo_g( GeoGpu* geo, int r, int ir, int ip ) {

  printf("start testGeo_g");
  GeoRegion*           regions = ( *geo ).regions;
  CaloDetDescrElement_Gpu* cells   = geo->cells;

  int                neta  = regions[r].cell_grid_eta();
  int                nphi  = regions[r].cell_grid_phi();
  unsigned long long index = regions[r].cell_grid_g()[ir * nphi + ip];
  printf( " From GPU.., region %d, cell_grid[%d][%d]: [%d][%d] index=%ld \n", r, ir, ip, neta, nphi, index );

  CaloDetDescrElement_Gpu* c = &cells[index];

  long long hashid = c->calo_hash();
  long long id     = c->identify();
  int       sample = c->getSampling();
  float     eta    = c->eta();
  float     phi    = c->phi();

  printf( " From GPU.., region %d, cell_grid[%d][%d]: index %lu index, hashid=%ld,eta=%f, phi=%f, sample=%d , ID=%ld "
          "cell_ptr=%#015lx \n",
          r, ir, ip, index, hashid, eta, phi, sample, id, regions[r].all_cells() );

  CaloDetDescrElement_Gpu cc = ( regions[r].all_cells() )[index];
  printf( " GPU test region have cells: cell index %llu, eta=%f phi=%f size of cell*GPU=%lu\n", index, cc.eta(),
          cc.phi(), sizeof( CaloDetDescrElement_Gpu* ) );
}

/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

bool GeoLoadGpu::TestGeo() {
  testGeo<<<1, 1>>>( m_cells_d, m_regions_d, m_ncells, m_nregions, 14, 0, 32 );
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) {
    std::cout << hipGetErrorString( err ) << std::endl;
    return false;
  }

  testGeo_g<<<1, 1>>>( m_geo_d, 14, 0, 32 );
  hipDeviceSynchronize();
  err = hipGetLastError();
  if ( err != hipSuccess ) {
    std::cout << hipGetErrorString( err ) << std::endl;
    return false;
  }

  long long* c   = m_regions[14].cell_grid();
  int        np  = m_regions[14].cell_grid_phi();
  int        ne  = m_regions[14].cell_grid_eta();
  int        idx = c[0 * np + 32];
  Identifier_Gpu Id  = m_cellid_array[idx];
  std::cout << "From Host: Region[14]Grid[0][32]: index=" << idx << ", ID=" << Id
            << ", HashCPU=" << ( *m_cells )[Id]->calo_hash() << ", neta=" << ne << ",  nphi=" << np
            << ", eta=" << ( *m_cells )[Id]->eta() << std::endl;

  return true;
  // end test
}

/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
bool GeoLoadGpu::SanityCheck() {
  // sanity check/test
  testHello<<<1, 1>>>();
  testCell<<<1, 1>>>( m_cells_d, 1872 );
  hipDeviceSynchronize();

  Identifier_Gpu Id = m_cellid_array[2000];

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) {
    std::cout << hipGetErrorString( err ) << std::endl;
    return false;
  }
  return true;
}

/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

bool GeoLoadGpu::LoadGpu_cu() {
  if ( !m_cells || m_ncells == 0 ) {
    std::cout << "Geometry is empty " << std::endl;
    return false;
  }

  GeoGpu geo_gpu_h;

  // Allocate Device memory for cells and copy cells as array
  // move cells on host to a array first
  if ( hipSuccess != hipMalloc( (void**)&m_cells_d, sizeof( CaloDetDescrElement_Gpu ) * m_ncells ) ) return false;

  CaloDetDescrElement_Gpu* cells_Host = (CaloDetDescrElement_Gpu*)malloc( m_ncells * sizeof( CaloDetDescrElement_Gpu ) );
  m_cellid_array                  = (Identifier_Gpu*)malloc( m_ncells * sizeof( Identifier_Gpu ) );

  // create an array of cell identities, they are in order of hashids.
  int ii = 0;
  for ( t_cellmap_Gpu::iterator ic = m_cells->begin(); ic != m_cells->end(); ++ic ) {
    cells_Host[ii]     = *( *ic ).second;
    Identifier_Gpu id      = ( ( *ic ).second )->identify();
    m_cellid_array[ii] = id;
    ii++;
  }

  if ( hipSuccess !=
       hipMemcpy( &m_cells_d[0], cells_Host, sizeof( CaloDetDescrElement_Gpu ) * m_ncells, hipMemcpyHostToDevice ) )
    return false;

  free( cells_Host );

  if ( 0 ) {
    if ( !SanityCheck() ) { return false; }
  }

  Rg_Sample_Index* SampleIndex_g;
  if ( hipSuccess != hipMalloc( (void**)&SampleIndex_g, sizeof( Rg_Sample_Index ) * m_max_sample ) ) return false;

  // copy sample_index array  to gpu
  if ( hipSuccess != hipMemcpy( SampleIndex_g, m_sample_index_h, sizeof( Rg_Sample_Index ) * m_max_sample,
                                  hipMemcpyHostToDevice ) ) {
    std::cout << "Error copy sample index " << std::endl;

    return false;
  }

  // each Region allocate a grid (long Long) gpu array
  //  copy array to GPU
  //  save to regions m_cell_g ;
  for ( unsigned int ir = 0; ir < m_nregions; ++ir ) {
    long long* ptr_g;
    if ( hipSuccess != hipMalloc( (void**)&ptr_g, sizeof( long long ) * m_regions[ir].cell_grid_eta() *
                                                        m_regions[ir].cell_grid_phi() ) )
      return false;
    if ( hipSuccess != hipMemcpy( ptr_g, m_regions[ir].cell_grid(),
                                    sizeof( long long ) * m_regions[ir].cell_grid_eta() * m_regions[ir].cell_grid_phi(),
                                    hipMemcpyHostToDevice ) )
      return false;
    m_regions[ir].set_cell_grid_g( ptr_g );
    m_regions[ir].set_all_cells( m_cells_d ); // set this so all region instance know where the GPU cells are, before
  }

  // GPU allocate Regions data  and load them to GPU as array of regions

  if ( hipSuccess != hipMalloc( (void**)&m_regions_d, sizeof( GeoRegion ) * m_nregions ) ) return false;
  if ( hipSuccess != hipMemcpy( m_regions_d, m_regions, sizeof( GeoRegion ) * m_nregions, hipMemcpyHostToDevice ) )
    return false;

  geo_gpu_h.cells        = m_cells_d;
  geo_gpu_h.ncells       = m_ncells;
  geo_gpu_h.nregions     = m_nregions;
  geo_gpu_h.regions      = m_regions_d;
  geo_gpu_h.max_sample   = m_max_sample;
  geo_gpu_h.sample_index = SampleIndex_g;

  // Now copy this to GPU and set the static member to this pointer
  hipMalloc( (void**)&m_geo_d, sizeof( GeoGpu ) );
  hipMemcpy( m_geo_d, &geo_gpu_h, sizeof( GeoGpu ), hipMemcpyHostToDevice );

  // more test for region grids
  if ( 0 ) { return TestGeo(); }
  return true;
}
