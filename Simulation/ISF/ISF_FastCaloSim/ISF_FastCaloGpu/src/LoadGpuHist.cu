#include "ISF_FastCaloGpu/LoadGpuHist.h"
#include <iostream>
#include "ISF_FastCaloGpu/gpuQ.h"
#include <hip/hip_runtime.h>

LoadGpuHist::LoadGpuHist() {}

LoadGpuHist::~LoadGpuHist() {
  free( m_hf1d );
  hipFree( ( *m_hf1d_h ).h_errors);
  hipFree( ( *m_hf1d_h ).h_contents );
  hipFree( ( *m_hf1d_h ).h_borders);
  free( m_hf1d_h );
  hipFree( m_hf1d_d );
}

void LoadGpuHist::LD1D() {
  // this call  assume  already have Histo set in m_hf
  // this function allocate memory of GPU and deep copy m_hf to m_hf_d
  if ( !m_hf1d ) {
    std::cout << "Error Load 1DHisto " << std::endl;
    return;
  }

  FH1D* hf_ptr = new FH1D;
  FH1D  hf;

  hf.nbins = ( *m_hf1d ).nbins;

  gpuQ( hipMalloc( (void**)&hf.h_borders, ( hf.nbins + 1 ) * sizeof( float ) ) );
  gpuQ( hipMalloc( (void**)&hf.h_contents,  hf.nbins  * sizeof( float ) ) );
  gpuQ( hipMalloc( (void**)&hf.h_errors,  hf.nbins  * sizeof( float ) ) );
  gpuQ( hipMemcpy( hf.h_borders, ( *m_hf1d ).h_borders, ( hf.nbins + 1 ) * sizeof( float ),
                    hipMemcpyHostToDevice ) );
  gpuQ( hipMemcpy( hf.h_contents, ( *m_hf1d ).h_contents,  hf.nbins  * sizeof( float ),
                    hipMemcpyHostToDevice ) );
  gpuQ( hipMemcpy( hf.h_errors, ( *m_hf1d ).h_errors,  hf.nbins  * sizeof( float ),
                    hipMemcpyHostToDevice ) );
  *( hf_ptr ) = hf;
  m_hf1d_h    = hf_ptr;

  gpuQ( hipMalloc( (void**)&m_hf1d_d, sizeof( FH1D ) ) );
  gpuQ( hipMemcpy( m_hf1d_d, m_hf1d_h, sizeof( FH1D ), hipMemcpyHostToDevice ) );
}

