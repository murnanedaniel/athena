#include "ISF_FastCaloGpu/gpuQ.h"
#include <iostream>

void gpu_assert( hipError_t code, const char* file, const int line ) {
  if ( code != hipSuccess ) {
    std::cerr << "gpu_assert: " << hipGetErrorString( code ) << " " << file << " " << line << std::endl;
    exit( code );
  }
}
