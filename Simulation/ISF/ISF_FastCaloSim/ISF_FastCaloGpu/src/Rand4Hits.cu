#include "ISF_FastCaloGpu/Rand4Hits.h"
#include "ISF_FastCaloGpu/gpuQ.h"
#include <iostream>
#include <hiprand.h>

#include "Rand4Hits_cpu.cxx"

#define CURAND_CALL( x )                                                                                               \
  if ( ( x ) != HIPRAND_STATUS_SUCCESS ) {                                                                              \
    printf( "Error at %s:%d\n", __FILE__, __LINE__ );                                                                  \
    exit( EXIT_FAILURE );                                                                                              \
  }

void Rand4Hits::allocate_simulation( long long /*maxhits*/, unsigned short /*maxbins*/, unsigned short maxhitct,
                                     unsigned long n_cells ) {

  float* Cells_Energy;
  int*   ct;
  gpuQ( hipMalloc( (void**)&Cells_Energy, n_cells * sizeof( float ) ) );
  m_cells_energy = Cells_Energy;
  Cell_E* cell_e;
  gpuQ( hipMalloc( (void**)&cell_e, maxhitct * sizeof( Cell_E ) ) );
  m_cell_e   = cell_e;
  m_cell_e_h = (Cell_E*)malloc( maxhitct * sizeof( Cell_E ) );
  gpuQ( hipMalloc( (void**)&ct, sizeof( int ) ) );
  m_ct = ct;
}

Rand4Hits::~Rand4Hits() {
  gpuQ( hipFree( m_rand_ptr ) );
  if ( m_useCPU ) {
    destroyCPUGen();
  } else {
    CURAND_CALL( hiprandDestroyGenerator( *( (hiprandGenerator_t*)m_gen ) ) );
    delete (hiprandGenerator_t*)m_gen;
  }
};

void Rand4Hits::rd_regen() {
  if ( m_useCPU ) {
    genCPU( 3 * m_total_a_hits );
    gpuQ( hipMemcpy( m_rand_ptr, m_rnd_cpu.data(), 3 * m_total_a_hits * sizeof( float ), hipMemcpyHostToDevice ) );
  } else {
    CURAND_CALL( hiprandGenerateUniform( *( (hiprandGenerator_t*)m_gen ), m_rand_ptr, 3 * m_total_a_hits ) );
  }
};

void Rand4Hits::create_gen( unsigned long long seed, size_t num, bool useCPU ) {

  printf( " number of random numbers : %zu with seed %llu \n", num, seed);
  float* f{nullptr};
  gpuQ( hipMalloc( &f, num * sizeof( float ) ) );

  m_useCPU = useCPU;
  
  if ( m_useCPU ) {
    createCPUGen( seed );
    genCPU( num );
    gpuQ( hipMemcpy( f, m_rnd_cpu.data(), num * sizeof( float ), hipMemcpyHostToDevice ) );
  } else {
    hiprandGenerator_t* gen = new hiprandGenerator_t;
    CURAND_CALL( hiprandCreateGenerator( gen, HIPRAND_RNG_PSEUDO_DEFAULT ) );
    CURAND_CALL( hiprandSetPseudoRandomGeneratorSeed( *gen, seed ) );
    CURAND_CALL( hiprandGenerateUniform( *gen, f, num ) );
    m_gen = (void*)gen;
  }

  m_rand_ptr = f;
}
