#include "hip/hip_runtime.h"
#include "ISF_FastCaloGpu/CaloGpuGeneral_cu.h"
#include "ISF_FastCaloGpu/GeoRegion.h"
#include "ISF_FastCaloGpu/GeoGpu_structs.h"
#include "ISF_FastCaloGpu/Hit.h"
#include "ISF_FastCaloGpu/Rand4Hits.h"

#include "ISF_FastCaloGpu/gpuQ.h"
#include "ISF_FastCaloGpu/Args.h"
#include <chrono>

#define BLOCK_SIZE 256

using namespace CaloGpuGeneral_fnc;

namespace CaloGpuGeneral_cu {

  __global__ void simulate_A( float E, int nhits, Chain0_Args args, bool reweight ) {

    long t = threadIdx.x + blockIdx.x * blockDim.x;
    if ( t < nhits ) {
      Hit hit;
      hit.E() = E;
      CenterPositionCalculation_d( hit, args );
      HistoLateralShapeParametrization_d( hit, t, args, reweight );
      HitCellMappingWiggle_d( hit, args, t );
    }
  }

  __global__ void simulate_ct( Chain0_Args args ) {

    unsigned long tid = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tid < args.ncells ) {
      if ( args.cells_energy[tid] > 0 ) {
        unsigned int ct = atomicAdd( args.hitcells_ct, 1 );
        Cell_E       ce;
        ce.cellid           = tid;
        ce.energy           = args.cells_energy[tid];
        args.hitcells_E[ct] = ce;
      }
    }
  }

  __global__ void simulate_clean( Chain0_Args args ) {
    unsigned long tid = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tid < args.ncells ) { args.cells_energy[tid] = 0.0; }
    if ( tid == 0 ) args.hitcells_ct[0] = 0;
  }

  __host__ void simulate_A_cu( float E, int nhits, Chain0_Args& args, bool reweight ) {
    int blocksize   = BLOCK_SIZE;
    int threads_tot = nhits;
    int nblocks     = ( threads_tot + blocksize - 1 ) / blocksize;
    simulate_A<<<nblocks, blocksize>>>( E, nhits, args, reweight );
  }

  __host__ void simulate_hits( float E, int nhits, Chain0_Args& args , bool reweight ) {

    hipError_t err = hipGetLastError();

    unsigned long ncells      = args.ncells;
    int           blocksize   = BLOCK_SIZE;
    int           threads_tot = args.ncells;
    int           nblocks     = ( threads_tot + blocksize - 1 ) / blocksize;

    // cleal the threads
    simulate_clean<<<nblocks, blocksize>>>( args );

    blocksize   = BLOCK_SIZE;
    threads_tot = nhits;
    nblocks     = ( threads_tot + blocksize - 1 ) / blocksize;

    // main cuda kernels
    // simulate the desposit energy for each hit
    simulate_A<<<nblocks, blocksize>>>( E, nhits, args, reweight );

    nblocks = ( ncells + blocksize - 1 ) / blocksize;
    // get the deposit energy
    simulate_ct<<<nblocks, blocksize>>>( args );

    int ct;
    gpuQ( hipMemcpy( &ct, args.hitcells_ct, sizeof( int ), hipMemcpyDeviceToHost ) );
    gpuQ( hipMemcpy( args.hitcells_E_h, args.hitcells_E, ct * sizeof( Cell_E ), hipMemcpyDeviceToHost ) );

    // pass result back
    args.ct = ct;
  }

} // namespace CaloGpuGeneral_cu
