// Dear emacs, this is -*- c++ -*-
//
// Copyright (C) 2002-2022 CERN for the benefit of the ATLAS collaboration
//

// Local include(s).
#include "KernelRunnerSvcImpl.h"

// AthCUDA include(s).
#include "AthCUDACore/Macros.cuh"
#include "AthCUDACore/StreamHolderHelpers.cuh"
#include "AthCUDACore/TaskArena.h"

#include "CxxUtils/checker_macros.h"

// System include(s).
#include <cassert>
#include <iostream>
#include <memory>

namespace {

#ifdef __HIPCC__
   /// Type used internally to signal the finish of kernel tasks
   class KernelFinishData {
   public:
      /// Constructor
      KernelFinishData( const AthCUDA::KernelRunnerSvcImplCallback& callback,
                        std::unique_ptr< AthCUDA::IKernelTask > task )
      : m_callback( callback ), m_task( std::move( task ) ) {}

      /// Callback to the kernel runner service
      AthCUDA::KernelRunnerSvcImplCallback m_callback;
      /// The task that's being executed
      std::unique_ptr< AthCUDA::IKernelTask > m_task;
   }; // struct KernelFinishData

   /// Function set up to be called by CUDA when a task finishes
   void setKernelFinished( void* userData ) {

      // Cast the user data to the right type.
      KernelFinishData* data =
         reinterpret_cast< KernelFinishData* >( userData );
      assert( data != nullptr );

      // Check if there was an error in the execution, and finish
      // the task accordingly.
      const int result =
         data->m_task->finished( hipGetLastError() == hipSuccess ? 0 : 1,
                                 AthCUDA::IKernelTask::Asynchronous );
      if( result != 0 ) {
         std::cerr << "Failed to finish a CUDA kernel task!" << std::endl;
         std::abort();
      }

      // Tell the service that a task has finished executing.
      data->m_callback();

      // Delete the data object.
      delete data;
      return;
   }
#endif // __HIPCC__

   /// Functor scheduling an @c AthCUDA::IKernelTask for execution
   // Thread-safety qualifiers are ok here, since this should be run
   // by only one thread at a time.
   class KernelSchedulerTask {

   public:
      /// Constructor with all necessary parameters
      KernelSchedulerTask( const AthCUDA::KernelRunnerSvcImplCallback& callback,
                           std::unique_ptr< AthCUDA::IKernelTask > task,
                           AthCUDA::KernelRunnerSvcImpl& svcImpl )
      : m_callback( callback ), m_task( std::move( task ) ),
        m_svcImpl( &svcImpl ) {}

      /// Operator executing the functor
      void operator()() const {

         // Get a stream for the job.
         AthCUDA::KernelRunnerSvcImpl* svcImpl ATLAS_THREAD_SAFE = m_svcImpl;
         auto stream = svcImpl->getAvailableStream();
         assert( stream );

         // First off, let the task schedule all of its own operations.
         if( m_task->execute( stream ) != 0 ) {
            std::cerr << "Failed to schedule the launch of a GPU task"
                      << std::endl;
            // At this point there's not much that the code can do. So let's
            // just sit back, and see how the job will eventually crash...
         }

         // Now add a step after those to the stream, one that signals to
         // us that the task is done.
         CUDA_EXP_CHECK( hipLaunchHostFunc( getStream( stream ),
                                             ::setKernelFinished,
                            new ::KernelFinishData( m_callback,
                                                    std::move( m_task ) ) ) );
      }

   private:
      /// Callback to the kernel runner service
      AthCUDA::KernelRunnerSvcImplCallback m_callback;
      /// The task that's being executed
      mutable std::unique_ptr< AthCUDA::IKernelTask > m_task ATLAS_THREAD_SAFE;
      /// Pointer to the service implementation object
      AthCUDA::KernelRunnerSvcImpl* m_svcImpl;

   }; // class KernelSchedulerTask

} // private namespace

namespace AthCUDA {

   void KernelRunnerSvcImpl::execute( std::unique_ptr< IKernelTask > task ) {

      // Schedule a task that will take care of scheduling/launching the CUDA
      // kernel.
      taskArena().enqueue( ::KernelSchedulerTask( m_callback,
                                                  std::move( task ),
                                                  *this ) );

      // Return gracefully.
      return;
   }

} // namespace AthCUDA
