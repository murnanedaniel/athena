#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <chrono>

#include "DataTypes.h"

//#include "KalmanFilter_kernel_ver2.cuh"
#include "KalmanFilter_kernel_ver3.cuh"


extern "C" float runKernel(INPUT_TRACK_INFO_TYPE* d_In,
         DETECTOR_SURFACE_TYPE* d_Geo,
         OUTPUT_TRACK_INFO_TYPE* d_Out,
         HIT_INFO_TYPE* d_Hit,int Ntracks) {

  int threadsPerBlock = BLOCK_SIZE;

  int blocksPerGrid;

  if(Ntracks<BLOCK_SIZE)
    blocksPerGrid=1;
  else
    blocksPerGrid = (Ntracks + threadsPerBlock - 1) / threadsPerBlock;

  hipDeviceSynchronize();

  auto kernelStart = std::chrono::steady_clock::now();

  KalmanFilterGPU<<<blocksPerGrid, threadsPerBlock>>>(d_In, d_Out, d_Geo, d_Hit, Ntracks);

  hipError_t err = hipGetLastError();

  if (err != hipSuccess) {
    printf("Kernel launch: error code : %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();

  auto kernelStop = std::chrono::steady_clock::now();

  return (float)std::chrono::duration_cast<std::chrono::nanoseconds>( kernelStop - kernelStart ).count() * 1E-6;
}
